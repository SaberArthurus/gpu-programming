// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}

__device__ float square_value (float a)
{
    return a * a;
}


__global__ void square_array (float *a, int n)
{
    int ind = threadIdx.x + blockDim.x * blockIdx.x;
    if (ind < n)
    {
        float val = a[ind];
        a[ind] = square_value(val);
    }
}



int main(int argc,char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 10;
    float *a = new float[n];
    for(int i=0; i<n; i++) a[i] = i;

    // CPU computation
    for(int i=0; i<n; i++)
    {
        float val = a[i];
        val = val*val;
        a[i] = val;
    }

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;
    


    // GPU computation
    // reinit data
    for(int i=0; i<n; i++) a[i] = i;

    
    // ###
    // ### TODO: Implement the "square array" operation on the GPU and store the result in "a"
    // ###
    // ### Notes:
    // ### 1. Remember to free all GPU arrays after the computation
    // ### 2. Always use the macro CUDA_CHECK after each CUDA call, e.g. "cudaMalloc(...); CUDA_CHECK;"
    // ###    For convenience this macro is defined directly in this file, later we will only include "aux.h"

    // initialize the array on GPU
    float *d_a = NULL;
    size_t nbytes = n * sizeof(float);
    hipMalloc(&d_a, nbytes); CUDA_CHECK;
    
    // move from host to device memory
    hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice); CUDA_CHECK;

    // initialize block and grid size
    dim3 block = dim3(2, 1, 1); 
    dim3 grid = dim3((n + block.x - 1) / block.x, 1, 1);

    // dispatch the kernel 
    square_array <<<grid, block>>> (d_a, n);


    // copy result back to host memory
    hipMemcpy(a, d_a, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;

    // free the device memory
    hipFree(d_a); CUDA_CHECK;

    // print result
    cout << "GPU:" << endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
}



