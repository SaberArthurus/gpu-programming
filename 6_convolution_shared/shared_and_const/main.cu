#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "aux.h"
#include <iostream>
#include <cmath>
using namespace std;
#include <cstdio>

#define _USE_MATH_DEFINES
// Standard deviation of the Gaussian
#define KERNEL_MAX_RADIUS 20

// Dimensions of the block
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 8
// uncomment to use the camera
// #define CAMERA
#define USE_CONST

__constant__ float constKernel[(2 * KERNEL_MAX_RADIUS + 1) * (2 * KERNEL_MAX_RADIUS + 1)];

__global__ void perform_convolution (float *d_imgIn, float *d_imgKern, float *d_imgOut, int w, int h, int nc, int r, int dim_share_x, int dim_share_y)
{
    // size_t global_x = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
    size_t block_start_x = blockIdx.x * blockDim.x;
    size_t block_start_y = blockIdx.y * blockDim.y;
    // int x = global_ind % w;
    // int y = global_ind % (w * h) / w;
    // int c = global_ind / (w * h);

    // -------------------------- INITIALIZE AND FILL SHARED MEMORY -----------------------------
    extern __shared__ float sh_imgIn[]; // will be of size (dim_share_x * dim_share_y * nc)

    for (int chan = 0; chan < nc; chan++)
    {
        for (int ind = threadIdx.x + blockDim.x * threadIdx.y; ind < dim_share_x * dim_share_y; ind += blockDim.x * blockDim.y)
        {
            // x and y indices within shared memory
            int sh_x = ind % dim_share_x;
            int sh_y = ind / dim_share_x;

            // x and y indices within the actual image
            int real_x = block_start_x - r + sh_x;
            int real_y = block_start_y - r + sh_y;

            // x and y adjusted not to overshoot the boundaries
            real_x = min(max(real_x, 0), w - 1);
            real_y = min(max(real_y, 0), h - 1);

            // copy to shared memory
            sh_imgIn[ind + dim_share_x * dim_share_y * chan] = d_imgIn[real_x + real_y * w + chan * w * h];
        }
    }

    // Ensure all threads have finished writing to shared memory
    __syncthreads();



    // ---------------------------- PERFORM THE COMPUTATION ---------------------------------------
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w and y < h)
    {
        for (int chan = 0; chan < nc; chan++)
        {
            float accumulated = 0;
            for (int a = -r; a <= r; a++)
            {
                for (int b = -r; b <= r; b++)
                {
                    int sh_x = x + a - block_start_x + r;
                    int sh_y = y + b - block_start_y + r;

#ifdef USE_CONST
                    accumulated += sh_imgIn[sh_x + sh_y * dim_share_x + chan * dim_share_x * dim_share_y]
                    * constKernel[(r + a) + (r + b) * (2 * r + 1)]; // Using kernel in constant memory
#else
                    accumulated += sh_imgIn[sh_x + sh_y * dim_share_x + chan * dim_share_x * dim_share_y]
                    * d_imgKern[(r + a) + (r + b) * (2 * r + 1)]; // Using the global kernel passed in as an argument
#endif
                }
            }
            d_imgOut[x + y * w + chan * w * h] = accumulated;

        }
    }
    __syncthreads();
}


int main(int argc, char **argv)
{
    // -------------------------- INITIALIZATION ----------------------------------
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;


    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // standard deviation of the Gaussian
    float sigma = 1;
    getParam("sigma", sigma, argc, argv);
    cout << "sigma: " << sigma << endl;
    sigma = abs(sigma);

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
    cv::VideoCapture camera(0);
    if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
    camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
    camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;


    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h, w, mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer






    // -------------------------- KERNEL COMPUTATION ---------------------------
    int r = (int)ceil(3 * sigma);
    int w_kernel = 2 * r + 1;
    int w_mid = r + 1;
    int h_mid = r + 1;
    cv::Mat mKernel = cv::Mat::zeros(w_kernel, w_kernel, CV_32FC1);

    // Normalize the kernel so that it sums up to 1
    float val = 0;

    for (int i = 0; i < w_kernel; i++)
    {
        for (int j = 0; j < w_kernel; j++)
        {
            val = 1.0 / (2.0 * M_PI * sigma * sigma) * exp(-(pow(i - w_mid, 2) + pow(j - h_mid, 2)) / (2.0 * sigma * sigma));
            mKernel.at<float>(i, j) = val;
        }
    }

    mKernel /= cv::sum(mKernel)[0];

    // Create kernel for visualization, that has max value of 1

    double minV, maxV;
    cv::Point minL, maxL;
    cv::minMaxLoc(mKernel, &minV, &maxV, &minL, &maxL);
    cv::Mat visKernel = mKernel / maxV;

    // Display the visualization kernel
    // showImage("Kernel", visKernel, 100 + w + 40,  100 + h + 40);  // show at position (x_from_left=100,y_from_above=100)
    // cv::waitKey(0);





    // ---------------------------- PREPARE THE ARRAYS  ON HOST ------------------------------
    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn  = new float[(size_t)w*h*nc];

    // allocate the linearized kernel array
    float *imgKern = new float[w_kernel * w_kernel];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);
    convert_mat_to_layered (imgKern, mKernel);
    




    // ------------------------------- PREPARE THE ARRAYS ON DEVICE ------------------------
    // Initialize the arrays on the device    
    float *d_imgIn = NULL;
    float *d_imgKern = NULL;
    float *d_imgOut = NULL;
    hipMalloc(&d_imgIn, w * h * nc * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgKern, w_kernel * w_kernel * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgOut, w * h * nc * sizeof(float)); CUDA_CHECK;

    // Constant kernel
    size_t kernel_bytes = w_kernel * w_kernel * sizeof(float);
    hipMemcpyToSymbol(HIP_SYMBOL(constKernel), imgKern, kernel_bytes);

    // Move input img and kernel to the device
    hipMemcpy(d_imgIn, imgIn, w * h * nc * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(d_imgKern, imgKern, w_kernel * w_kernel * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    




    // ------------------------------- DISPATCH THE KERNELS ---------------------------------
    dim3 block = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1); 
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);

    size_t dim_share_x = BLOCK_SIZE_X + 2 * r;
    size_t dim_share_y = BLOCK_SIZE_Y + 2 * r;
    size_t size_share_bytes = dim_share_x * dim_share_y * nc * sizeof(float);

    Timer timer; timer.start();
    for (int i = 0; i < repeats; i++)
    {    
        perform_convolution <<<grid, block, size_share_bytes>>> (d_imgIn, d_imgKern, d_imgOut, w, h, nc, r, dim_share_x, dim_share_y);
    }
    timer.end();  float t = timer.get();  // elapsed time in seconds
#ifdef USE_CONST    
    cout << "Average time over " << repeats << " runs using kernel in const: " << t * 1000.0 / repeats << " ms" << endl;
#else
    cout << "Average time over " << repeats << " runs using kernel in shared: " << t * 1000.0 / repeats << " ms" << endl;
#endif


    // ------------------------------ COLLECT DATA AND CLEAN UP ---------------------------------
    hipMemcpy(imgOut, d_imgOut, w * h * nc * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgKern); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif


    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



